#include "hip/hip_runtime.h"
// CS6354: Computer Architecture
// GPU Assignment: Find the maximum value in a big vector
// Apr. 8, 2014
// Copyright (c) 2021 Deyuan Guo <guodeyuan@gmail.com>. All rights reserved.

#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <cfloat>

// VERSION 1.3 MODIFIED 4/2 11:32PM by Jack

// The number of threads per blocks in the kernel
// (if we define it here, then we can use its value in the kernel,
//  for example to statically declare an array in shared memory)
const int threads_per_block = 256;


// Forward function declarations
float GPU_vector_max(float *A, int N, int kernel_code);
float CPU_vector_max(float *A, int N);
float *get_random_vector(int N);
float *get_increasing_vector(int N);
long long start_timer();
long long stop_timer(long long start_time, char *name);
void die(char *message);
void checkError();

int main(int argc, char **argv) {

    //default kernel
    int kernel_code = 1;

    // Parse vector length and kernel options
    int N;
    if(argc == 2) {
        N = atoi(argv[1]); // user-specified value
    } else if (argc == 4 && !strcmp(argv[2], "-k")) {
        N = atoi(argv[1]); // user-specified value
        kernel_code = atoi(argv[3]);
        printf("KERNEL_CODE %d\n", kernel_code);
    } else {
        die("USAGE: ./vector_max <vector_length> -k <kernel_code>");
    }

    // Seed the random generator (use a constant here for repeatable results)
    srand(10);

    // Generate a random vector
    // You can use "get_increasing_vector()" for debugging
    long long vector_start_time = start_timer();
    float *vec = get_random_vector(N);
    //float *vec = get_increasing_vector(N);
    stop_timer(vector_start_time, "Vector generation");

    // Compute the max on the GPU
    long long GPU_start_time = start_timer();
    float result_GPU = GPU_vector_max(vec, N, kernel_code);
    long long GPU_time = stop_timer(GPU_start_time, "\t            Total");

    // Compute the max on the CPU
    long long CPU_start_time = start_timer();
    float result_CPU = CPU_vector_max(vec, N);
    long long CPU_time = stop_timer(CPU_start_time, "\nCPU");

    // Free vector
    free(vec);

    // Compute the speedup or slowdown
    if (GPU_time > CPU_time) printf("\nCPU outperformed GPU by %.2fx\n", (float) GPU_time / (float) CPU_time);
    else                     printf("\nGPU outperformed CPU by %.2fx\n", (float) CPU_time / (float) GPU_time);

    // Check the correctness of the GPU results
    int wrong = result_CPU != result_GPU;

    // Report the correctness results
    if(wrong) printf("GPU output %f did not match CPU output %f\n", result_GPU, result_CPU);

}


// A GPU kernel that computes the maximum value of a vector
// (each lead thread (threadIdx.x == 0) computes a single value
__global__ void vector_max_kernel(float *in, float *out, int N) {

    // Determine the "flattened" block id and thread id
    int block_id = blockIdx.x + gridDim.x * blockIdx.y;
    int thread_id = blockDim.x * block_id + threadIdx.x;

    // A single "lead" thread in each block finds the maximum value over a range of size threads_per_block
    float max = 0.0;
    if (threadIdx.x == 0) {

        //calculate out of bounds guard
        //our block size will be 256, but our vector may not be a multiple of 256!
        int end = threads_per_block;
        if(thread_id + threads_per_block > N)
            end = N - thread_id;

        //grab the lead thread's value
        max = in[thread_id];

        //grab values from all other threads' locations
        for(int i = 1; i < end; i++) {

            //if larger, replace
            if(max < in[thread_id + i])
                max = in[thread_id + i];
        }

        //out[blockIdx.x] = max;
        out[block_id] = max;

    }
}

// A GPU kernel that computes the maximum value of a vector
// (each lead thread (threadIdx.x == 0) computes a single value
__global__ void vector_max_kernel2(float *in, float *out, int N) {

    // Determine the "flattened" block id and thread id
    int block_id = blockIdx.x + gridDim.x * blockIdx.y;
    int thread_id = blockDim.x * block_id + threadIdx.x;

    // Load numbers to shared memory and sync threads.
    __shared__ float sh_array[threads_per_block];
    if (thread_id < N) sh_array[threadIdx.x] = in[thread_id];
    else sh_array[threadIdx.x] = 0.0;
    __syncthreads();

    // A single "lead" thread in each block finds the maximum value over a range of size threads_per_block
    if (threadIdx.x == 0) {
        //grab the lead thread's value
        float max = sh_array[0];

        //grab values from all other threads' locations
        for (int i = 1; i < threads_per_block; i++) {
            //if larger, replace
            if (max < sh_array[i])
                max = sh_array[i];
        }

        // The max value in current block.
        out[block_id] = max;
    }
}

// A GPU kernel that computes the maximum value of a vector
// (each lead thread (threadIdx.x == 0) computes a single value
__global__ void vector_max_kernel3(float *in, float *out, int N) {

    // Determine the "flattened" block id and thread id
    int block_id = blockIdx.x + gridDim.x * blockIdx.y;
    int thread_id = blockDim.x * block_id + threadIdx.x;

    // Load numbers to shared memory and sync threads.
    __shared__ float sh_array[threads_per_block];
    if (thread_id < N) sh_array[threadIdx.x] = in[thread_id];
    else sh_array[threadIdx.x] = 0.0;
    __syncthreads();

    // Interleaved binary reduction.
    for (int i = 1; i < threads_per_block; i += i) {
        int idx1 = threadIdx.x;
        int idx2 = threadIdx.x + i;
        if (idx1 % (i + i) == 0) {
            if (sh_array[idx1] < sh_array[idx2])
                sh_array[idx1] = sh_array[idx2];
        }
        __syncthreads();
    }

    // The max value in current block.
    out[block_id] = sh_array[0];
}

// A GPU kernel that computes the maximum value of a vector
// (each lead thread (threadIdx.x == 0) computes a single value
__global__ void vector_max_kernel4(float *in, float *out, int N) {

    // Determine the "flattened" block id and thread id
    int block_id = blockIdx.x + gridDim.x * blockIdx.y;
    int thread_id = blockDim.x * block_id + threadIdx.x;

    // Load numbers to shared memory and sync threads.
    __shared__ float sh_array[threads_per_block];
    if (thread_id < N) sh_array[threadIdx.x] = in[thread_id];
    else sh_array[threadIdx.x] = 0.0;
    __syncthreads();

    // Binary reduction.
    for (int i = threads_per_block >> 1; i > 0; i >>= 1) {
        int idx1 = threadIdx.x;
        int idx2 = threadIdx.x + i;
        if (idx1 < i) {
            if (sh_array[idx1] < sh_array[idx2])
                sh_array[idx1] = sh_array[idx2];
        }
        __syncthreads();
    }

    // The max value in current block.
    out[block_id] = sh_array[0];
}

// Returns the maximum value within a vector of length N
float GPU_vector_max(float *in_CPU, int N, int kernel_code) {

    int vector_size = N * sizeof(float);

    // Allocate CPU memory for the result
    float *out_CPU = (float *) malloc(vector_size);
    if (out_CPU == NULL) die("Error allocating CPU memory");

    // Allocate GPU memory for the inputs and the result
    long long memory_start_time = start_timer();

    float *in_GPU, *out_GPU;
    if (hipMalloc((void **) &in_GPU, vector_size) != hipSuccess) die("Error allocating GPU memory");
    if (hipMalloc((void **) &out_GPU, vector_size) != hipSuccess) die("Error allocating GPU memory");

    // Transfer the input vectors to GPU memory
    hipMemcpy(in_GPU, in_CPU, vector_size, hipMemcpyHostToDevice);

    stop_timer(memory_start_time, "\nGPU:\t  Transfer to GPU");

    // Execute the kernel to compute the vector sum on the GPU
    long long kernel_start_time;
    kernel_start_time = start_timer();

    // Problem 1: Iterately call the kernel function.
    do {
        // Determine the number of thread blocks in the x- and y-dimension
        int num_blocks = (int) ((float) (N + threads_per_block - 1) / (float) threads_per_block);
        int max_blocks_per_dimension = 65535;
        int num_blocks_y = (int) ((float) (num_blocks + max_blocks_per_dimension - 1) / (float) max_blocks_per_dimension);
        int num_blocks_x = (int) ((float) (num_blocks + num_blocks_y - 1) / (float) num_blocks_y);
        dim3 grid_size(num_blocks_x, num_blocks_y, 1);

        //printf("# N = %d, threads_per_block = %d, num_blocks = %d, num_blocks_y = %d, num_blocks_x = %d\n", N, threads_per_block, num_blocks, num_blocks_y, num_blocks_x);

        switch(kernel_code){
        case 1 :
            vector_max_kernel <<< grid_size , threads_per_block >>> (in_GPU, out_GPU, N);
            break;
        case 2 :
            //LAUNCH KERNEL FROM PROBLEM 2 HERE
            vector_max_kernel2 <<< grid_size , threads_per_block >>> (in_GPU, out_GPU, N);
            break;
        case 3 :
            //LAUNCH KERNEL FROM PROBLEM 3 HERE
            vector_max_kernel3 <<< grid_size , threads_per_block >>> (in_GPU, out_GPU, N);
            break;
        case 4 :
            //LAUNCH KERNEL FROM PROBLEM 4 HERE
            vector_max_kernel4 <<< grid_size , threads_per_block >>> (in_GPU, out_GPU, N);
            break;
        default :
            die("INVALID KERNEL CODE\n");
        }

        // New value of N.
        N = num_blocks;
        // If there is only one max value, then stop.
        if (N <= 1) break;

        // Swap in_GPU and out_GPU to reuse the space.
        float * tmp = in_GPU; in_GPU = out_GPU; out_GPU = tmp;
    } while (1);


    hipDeviceSynchronize();  // this is only needed for timing purposes
    stop_timer(kernel_start_time, "\t Kernel execution");

    checkError();

    // Transfer the result from the GPU to the CPU
    memory_start_time = start_timer();

    //copy C back
    //hipMemcpy(out_CPU, out_GPU, vector_size, hipMemcpyDeviceToHost);
    hipMemcpy(out_CPU, out_GPU, sizeof(float), hipMemcpyDeviceToHost);
    checkError();
    stop_timer(memory_start_time, "\tTransfer from GPU");

    // Free the GPU memory
    hipFree(in_GPU);
    hipFree(out_GPU);

    float max = out_CPU[0];
    free(out_CPU);

    //return a single statistic
    return max;
}


// Returns the maximum value within a vector of length N
float CPU_vector_max(float *vec, int N) {

    // find the max
    float max;
    max = vec[0];
    for (int i = 1; i < N; i++) {
        if(max < vec[i]) {
            max = vec[i];
        }
    }

    // Return a single statistic
    return max;
}


// Returns a randomized vector containing N elements
float *get_random_vector(int N) {
    if (N < 1) die("Number of elements must be greater than zero");

    // Allocate memory for the vector
    float *V = (float *) malloc(N * sizeof(float));
    if (V == NULL) die("Error allocating CPU memory");

    // Populate the vector with random numbers
    for (int i = 0; i < N; i++) V[i] = (float) rand() / (float) rand();

    // Return the randomized vector
    return V;
}

float *get_increasing_vector(int N) {
    if (N < 1) die("Number of elements must be greater than zero");

    // Allocate memory for the vector
    float *V = (float *) malloc(N * sizeof(float));
    if (V == NULL) die("Error allocating CPU memory");

    // Populate the vector with random numbers
    for (int i = 0; i < N; i++) V[i] = (float) i;

    // Return the randomized vector
    return V;
}

void checkError() {
    // Check for kernel errors
    hipError_t error = hipGetLastError();
    if (error) {
        char message[256];
        sprintf(message, "CUDA error: %s", hipGetErrorString(error));
        die(message);
    }
}

// Returns the current time in microseconds
long long start_timer() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec * 1000000 + tv.tv_usec;
}


// Prints the time elapsed since the specified time
long long stop_timer(long long start_time, char *name) {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
    printf("%s: %.5f sec\n", name, ((float) (end_time - start_time)) / (1000 * 1000));
    return end_time - start_time;
}


// Prints the specified message and quits
void die(char *message) {
    printf("%s\n", message);
    exit(1);
}
